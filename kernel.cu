#include "hip/hip_runtime.h"
// Ĭ����h��ͷ��ʾ�����ϵı�������d��ͷ��ʾ�豸�ϵı���
#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


//˫������ӳ���
__global__ void gpu_add(int d_a, int d_b, int* d_c) {
	// �ú������������α���Ϊ���룬�����ӷ��洢�ڵ���������ָ��d_cָ����ڴ�λ��
	// �豸��������ֵΪvoid����Ϊ����������洢��ָ��ָ����ڴ��У���������ʽ�ط����κ�ֵ
	*d_c = d_a + d_b;
}


int main(void) {
	// �������������Դ洢���
	int h_c;
	// �����豸ָ��
	int* d_c;
	// Ϊ�豸ָ������ڴ�
	hipMalloc((void**)&d_c, sizeof(int));  // �ú�������malloc�������������豸�Ϸ���d_c�������ڴ�
	// ����1��4��Ϊ���벢������洢��d_c���������ں˵���
	// << <1, 1> >> ��ʾִ��1��block��ÿ��blockһ���߳�
	gpu_add << <1, 1 >> > (1, 4, d_c);
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("1 + 4 = %d\n", h_c);
	// �ͷ��豸�ϵ��ڴ�
	hipFree(d_c);
	return 0;
}